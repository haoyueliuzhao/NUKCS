#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <corecrt_math_defines.h>
#include <>

using namespace std;
using namespace std::chrono;

// CUDA 核函数来计算 FFT
__global__ void fftKernel(hipDoubleComplex* d_A, hipDoubleComplex* d_fft, int x) {
    extern __shared__ hipDoubleComplex shared[];
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < x) {
        hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
        for (int n = 0; n < x; ++n) {
            double theta = -2.0 * M_PI * k * n / x;
            hipDoubleComplex exp = make_hipDoubleComplex(cos(theta), sin(theta));
            sum = hipCadd(sum, hipCmul(d_A[n], exp));
        }
        d_fft[k] = sum;
    }
}

// CUDA 核函数来计算 IFFT
__global__ void ifftKernel(hipDoubleComplex* d_B, hipDoubleComplex* d_ifft, int x) {
    extern __shared__ hipDoubleComplex shared[];
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < x) {
        hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
        for (int n = 0; n < x; ++n) {
            double theta = 2.0 * M_PI * k * n / x;
            hipDoubleComplex exp = make_hipDoubleComplex(cos(theta), sin(theta));
            sum = hipCadd(sum, hipCmul(d_B[n], exp));
        }
        d_ifft[k] = make_hipDoubleComplex(hipCreal(sum) / x, hipCimag(sum) / x);
    }
}

int main() {
    const int numExperiments = 5; // 每个规模的实验次数
    const int initialSize = 128;
    int n = 64;
    int m = 64;

    for (int i = 0; i < 10; i++) {
        n = n * 2;
        m = m * 2;
        int y = 1;

        // 计算y，使其为2的幂次
        while (y < n + m + 1) {
            y *= 2;
        }

        double totalTime = 0.0;

        for (int j = 0; j < numExperiments; j++) {
            // 生成随机测试用例
            vector<hipDoubleComplex> N;
            vector<hipDoubleComplex> M;
            srand(time(0)); // 随机数种子
            for (int k = 0; k <= n; k++) {
                int x = rand() % 10 + 1;
                hipDoubleComplex temp = make_hipDoubleComplex(x, 0);
                N.push_back(temp);
            }
            reverse(N.begin(), N.end());
            for (int k = n + 1; k < y; k++) {
                N.push_back(make_hipDoubleComplex(0, 0));
            }

            for (int k = 0; k <= m; k++) {
                int x = rand() % 10 + 1;
                hipDoubleComplex temp = make_hipDoubleComplex(x, 0);
                M.push_back(temp);
            }
            reverse(M.begin(), M.end());
            for (int k = m + 1; k < y; k++) {
                M.push_back(make_hipDoubleComplex(0, 0));
            }

            // 分配设备内存
            hipDoubleComplex* d_N;
            hipDoubleComplex* d_M;
            hipDoubleComplex* d_fft_N;
            hipDoubleComplex* d_fft_M;
            hipDoubleComplex* d_fft_Result;
            hipDoubleComplex* d_Result;

            hipMalloc((void**)&d_N, y * sizeof(hipDoubleComplex));
            hipMalloc((void**)&d_M, y * sizeof(hipDoubleComplex));
            hipMalloc((void**)&d_fft_N, y * sizeof(hipDoubleComplex));
            hipMalloc((void**)&d_fft_M, y * sizeof(hipDoubleComplex));
            hipMalloc((void**)&d_fft_Result, y * sizeof(hipDoubleComplex));
            hipMalloc((void**)&d_Result, y * sizeof(hipDoubleComplex));

            // 复制数据到设备
            hipMemcpy(d_N, N.data(), y * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
            hipMemcpy(d_M, M.data(), y * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

            auto startFFT = high_resolution_clock::now();

            // 执行 FFT 核函数
            int threadsPerBlock = 256;
            int blocksPerGrid = (y + threadsPerBlock - 1) / threadsPerBlock;
            fftKernel << <blocksPerGrid, threadsPerBlock, y * sizeof(hipDoubleComplex) >> > (d_N, d_fft_N, y);
            fftKernel << <blocksPerGrid, threadsPerBlock, y * sizeof(hipDoubleComplex) >> > (d_M, d_fft_M, y);
            hipDeviceSynchronize();

            auto endFFT = high_resolution_clock::now();
            duration<double> timeFFT = endFFT - startFFT;

            // 复制 FFT 结果回主机
            vector<hipDoubleComplex> fft_N(y);
            vector<hipDoubleComplex> fft_M(y);
            hipMemcpy(fft_N.data(), d_fft_N, y * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
            hipMemcpy(fft_M.data(), d_fft_M, y * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

            // 计算结果的点乘
            vector<hipDoubleComplex> fft_Result(y);
            for (int k = 0; k < y; k++) {
                fft_Result[k] = hipCmul(fft_N[k], fft_M[k]);
            }

            // 复制点乘结果到设备
            hipMemcpy(d_fft_Result, fft_Result.data(), y * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

            auto startIFFT = high_resolution_clock::now();

            // 执行 IFFT 核函数
            ifftKernel << <blocksPerGrid, threadsPerBlock, y * sizeof(hipDoubleComplex) >> > (d_fft_Result, d_Result, y);
            hipDeviceSynchronize();

            auto endIFFT = high_resolution_clock::now();
            duration<double> timeIFFT = endIFFT - startIFFT;

            // 复制 IFFT 结果回主机
            vector<hipDoubleComplex> Result(y);
            hipMemcpy(Result.data(), d_Result, y * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

            totalTime += (timeFFT + timeIFFT).count();

            // 释放设备内存
            hipFree(d_N);
            hipFree(d_M);
            hipFree(d_fft_N);
            hipFree(d_fft_M);
            hipFree(d_fft_Result);
            hipFree(d_Result);
        }

        double averageTime = totalTime / numExperiments;
        cout << "n=" << n << ", m=" << m << endl;
        cout << "平均总时间: " << averageTime * 1000 << " 毫秒" << endl;
        cout << endl;
    }

    return 0;
}
